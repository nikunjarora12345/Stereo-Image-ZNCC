#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <chrono>
#include <cassert>
#include <vector>

#include "lodepng.h"

hipError_t status;
#define CudaCall(x) \
	status = x; \
	if (status != hipSuccess) \
		std::cout << "Error [" << status << "]: " << hipGetErrorString(status) << " (" << __FILE__ << ":" << __LINE__ << ")" << std::endl


// GPU Kernel functions
#pragma region gpuCode

__global__
void ScaleAndGray(unsigned char* orig, unsigned* gray, unsigned width, unsigned height, int scaleFactor) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= height || j >= width)
		return;

	int newWidth = width / scaleFactor;

	int x = (scaleFactor * i - 1 * (i > 0));
	int y = (scaleFactor * j - 1 * (j > 0));

	gray[i * newWidth + j] =
		0.3 * orig[x * (4 * width) + 4 * y] +
		0.59 * orig[x * (4 * width) + 4 * y + 1] +
		0.11 * orig[x * (4 * width) + 4 * y + 2];
}

__global__
void Zncc(unsigned* leftPixels, unsigned* rightPixels, unsigned* disparityMap, unsigned width, unsigned height,
	int minDisp, int maxDisp, int windowWidth, int windowHeight) {
	
	unsigned windowSize = windowWidth * windowHeight;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= height || j >= width)
		return;

	float bestDisparity = maxDisp;
	float bestZncc = -1;

	// Select the best disparity value for the current pixel
	for (int d = minDisp; d <= maxDisp; d++) {
		// Calculating mean of blocks using the sliding window method
		float meanLBlock = 0, meanRBlock = 0;

		for (int x = -windowHeight / 2; x < windowHeight / 2; x++) {
			for (int y = -windowWidth / 2; y < windowWidth / 2; y++) {
				// Check for image borders
				if (
					!(i + x >= 0) ||
					!(i + x < height) ||
					!(j + y >= 0) ||
					!(j + y < width) ||
					!(j + y - d >= 0) ||
					!(j + y - d < width)
					) {
					continue;
				}

				meanLBlock += leftPixels[(i + x) * width + (j + y)];
				meanRBlock += rightPixels[(i + x) * width + (j + y - d)];
			}
		}

		meanLBlock /= windowSize;
		meanRBlock /= windowSize;

		// Calculate ZNCC for current disparity value
		float stdLBlock = 0, stdRBlock = 0;
		float currentZncc = 0;

		for (int x = -windowHeight / 2; x < windowHeight / 2; x++) {
			for (int y = -windowWidth / 2; y < windowWidth / 2; y++) {
				// Check for image borders
				if (
					!(i + x >= 0) ||
					!(i + x < height) ||
					!(j + y >= 0) ||
					!(j + y < width) ||
					!(j + y - d >= 0) ||
					!(j + y - d < width)
					) {
					continue;
				}

				int centerL = leftPixels[(i + x) * width + (j + y)] - meanLBlock;
				int centerR = rightPixels[(i + x) * width + (j + y - d)] - meanRBlock;

				// standard deviation
				stdLBlock += centerL * centerL;
				stdRBlock += centerR * centerR;

				currentZncc += centerL * centerR;
			}
		}

		currentZncc /= sqrtf(stdLBlock) * sqrtf(stdRBlock);

		// Selecting best disparity
		if (currentZncc > bestZncc) {
			bestZncc = currentZncc;
			bestDisparity = d;
		}
	}

	disparityMap[i * width + j] = (unsigned)fabs(bestDisparity);
}

__global__
void CrossCheck(unsigned* leftDisp, unsigned* rightDisp, unsigned* result, unsigned imSize, int crossCheckingThreshold) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= imSize)
		return;

	int diff = leftDisp[i] - rightDisp[i];
	if (diff >= 0) { // leftDisp is winner
		if (diff <= crossCheckingThreshold) {
			result[i] = leftDisp[i];
		} else {
			result[i] = 0;
		}
	} else { //  rightDisp is winner
		if (-diff <= crossCheckingThreshold) {
			result[i] = rightDisp[i];
		} else {
			result[i] = 0;
		}

	}
}

__global__
void OcclusionFill(unsigned* map, unsigned* result, unsigned width, unsigned height, int occlusionNeighbours) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= height || j >= width)
		return;

	unsigned currentIndex = i * width + j;
	result[currentIndex] = map[currentIndex];

	// If the pixel value is 0, copy value from nearest non zero neighbour
	if (map[currentIndex] == 0) {
		bool stop = false;

		for (int n = 1; n <= occlusionNeighbours / 2 && !stop; n++) {
			for (int y = -n; y <= n && !stop; y++) {
				for (int x = -n; x <= n && !stop; x++) {
					// Checking for borders
					if (
						!(i + x >= 0) ||
						!(i + x < height) ||
						!(j + y >= 0) ||
						!(j + y < width) ||
						(x == 0 && y == 0)
						) {
						continue;
					}

					int index = (i + x) * width + (j + y);

					if (map[index] == 0) {
						result[currentIndex] = map[index];
						stop = true;
						break;
					}
				}
			}
		}
	}
}

#pragma endregion gpuCode


/*
Class to calculate time taken by functions in seconds.
* Creating an object of the class in a function, calls the constructor which starts the timer.
* At the end of the function, the destructor is called which stops the timer and calculates the duration.
* We can get the duration manually using the getElapsedTime method.
*/
class Timer {
private:
	std::chrono::time_point<std::chrono::steady_clock> m_Start, m_End;
	std::chrono::duration<float> m_Duration;

public:
	Timer() {
		m_Start = std::chrono::high_resolution_clock::now();
	}

	~Timer() {
		m_End = std::chrono::high_resolution_clock::now();
		m_Duration = m_End - m_Start;

		std::cout << "Done (" << m_Duration.count() << " s)" << std::endl;
	}

	float getElapsedTime() {
		m_End = std::chrono::high_resolution_clock::now();
		m_Duration = m_End - m_Start;

		return m_Duration.count();
	}
};

constexpr int maxDisparity = 64;

constexpr int windowWidth = 15;
constexpr int windowHeight = 15;

constexpr int crossCheckingThreshold = 2;

constexpr int occlusionNeighbours = 256;

constexpr int scaleFactor = 4;

std::vector<unsigned char> loadImage(const char*, unsigned&, unsigned&);
std::vector<unsigned char> normalize(std::vector<unsigned>, const unsigned, const unsigned);

// Display GPU info
// https://stackoverflow.com/a/5689133
void DisplayHeader() {
	const int kb = 1024;
	const int mb = kb * kb;
	std::cout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

	std::cout << "CUDA version:   v" << CUDART_VERSION << std::endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	std::cout << "CUDA Devices: " << std::endl << std::endl;

	for (int i = 0; i < devCount; ++i) {
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		std::cout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
		std::cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << std::endl;
		std::cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << std::endl;
		std::cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << std::endl;
		std::cout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

		std::cout << "  Warp size:         " << props.warpSize << std::endl;
		std::cout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
		std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
		std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << std::endl;
		std::cout << std::endl;
	}
}

int main() {
	Timer timer;

	DisplayHeader();

	// Host variables
	std::vector<unsigned char> leftPixels, rightPixels;
	unsigned width, height, rightWidth, rightHeight;

	std::cout << "Reading Left Image...";
	leftPixels = loadImage("imageL.png", width, height);

	std::cout << "Reading Right Image...";
	rightPixels = loadImage("imageR.png", rightWidth, rightHeight);

	// left and right images are assumed to be of same dimensions
	assert(width == rightWidth && height == rightHeight);

	width /= scaleFactor;
	height /= scaleFactor;

	unsigned imSize = width * height;
	unsigned origSize = rightWidth * rightHeight;
	std::vector<unsigned> output(imSize);

	// Device variabels
	unsigned char *d_origL, *d_origR;
	unsigned *d_grayL, *d_grayR, *d_dispLR, *d_dispRL, *d_dispCC, *d_output;

	CudaCall(hipMalloc((void**) &d_origL, sizeof(unsigned char) * origSize * 4));
	CudaCall(hipMalloc((void**) &d_origR, sizeof(unsigned char) * origSize * 4));
	CudaCall(hipMalloc((void**) &d_grayL, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**) &d_grayR, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**) &d_dispLR, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**) &d_dispRL, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**) &d_dispCC, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**) &d_output, sizeof(unsigned) * imSize));

	// Copy Data from host to device
	CudaCall(hipMemcpy(d_origL, leftPixels.data(), sizeof(leftPixels[0]) * leftPixels.size(), hipMemcpyHostToDevice));
	CudaCall(hipMemcpy(d_origR, rightPixels.data(), sizeof(rightPixels[0]) * rightPixels.size(), hipMemcpyHostToDevice));

	// Profiling
	float elapsed = 0;
	hipEvent_t start, stop;

	CudaCall(hipEventCreate(&start));
	CudaCall(hipEventCreate(&stop));

	// Kernel Calls
	dim3 blocks(height / 21, width / 21);
	dim3 threads(21, 21);
	dim3 blocks1D((height / 21) * (width / 21));
	dim3 threads1D(21 * 21);

	// Scale and Gray left
	std::cout << "Converting Left Image to grayscale...";
	CudaCall(hipEventRecord(start));
	
	ScaleAndGray<<<blocks, threads>>>(d_origL, d_grayL, rightWidth, rightHeight, scaleFactor);
	
	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Scale and Gray right
	std::cout << "Converting Right Image to grayscale...";
	CudaCall(hipEventRecord(start));

	ScaleAndGray<<<blocks, threads>>>(d_origR, d_grayR, rightWidth, rightHeight, scaleFactor);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Disparity Left over Right
	std::cout << "Converting Left Disparity Map...";
	CudaCall(hipEventRecord(start));

	Zncc<<<blocks, threads>>>(d_grayL, d_grayR, d_dispLR, width, height, 0, maxDisparity, windowWidth, windowHeight);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Disparity Right over Left
	std::cout << "Converting Right Disparity Map...";
	CudaCall(hipEventRecord(start));

	Zncc<<<blocks, threads>>>(d_grayR, d_grayL, d_dispRL, width, height, -maxDisparity, 0, windowWidth, windowHeight);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Cross Checking
	std::cout << "Performing Cross Checking...";
	CudaCall(hipEventRecord(start));

	CrossCheck<<<blocks1D, threads1D>>>(d_dispLR, d_dispRL, d_dispCC, imSize, crossCheckingThreshold);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Occlusion Filling
	std::cout << "Performing Occlusion Filling...";
	CudaCall(hipEventRecord(start));

	OcclusionFill<<<blocks, threads>>>(d_dispCC, d_output, width, height, occlusionNeighbours);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Copy data from device to host
	CudaCall(hipMemcpy(&output[0], d_output, sizeof(unsigned) * imSize, hipMemcpyDeviceToHost));

	lodepng::encode("output.png", normalize(output, width, height), width, height);

	std::cout << "The program took " << timer.getElapsedTime() << " s" << std::endl;

	hipFree(d_origL);
	hipFree(d_origR);
	hipFree(d_grayL);
	hipFree(d_grayR);
	hipFree(d_dispLR);
	hipFree(d_dispRL);
	hipFree(d_dispCC);
	hipFree(d_output);

	std::cin.get();
	return 0;
}

std::vector<unsigned char> loadImage(const char* filename, unsigned& width, unsigned& height) {
	Timer timer;

	std::vector<unsigned char> pixels;

	unsigned error = lodepng::decode(pixels, width, height, filename);
	if (error) {
		std::cout << "Failed to load image: " << lodepng_error_text(error) << std::endl;
		std::cin.get();
		exit(-1);
	}

	return pixels;
}

std::vector<unsigned char> normalize(
	std::vector<unsigned> in,
	const unsigned width,
	const unsigned height
) {
	std::vector<unsigned char> result(width * height * 4);

	unsigned char max = 0;
	unsigned char min = UCHAR_MAX;

	for (int i = 0; i < width * height; i++) {
		if (in[i] > max) {
			max = in[i];
		}

		if (in[i] < min) {
			min = in[i];
		}
	}

	// Normalize values to be between 0 and 255
	int mapIndex = 0;
	for (int i = 0; i < width * height * 4; i += 4, mapIndex++) {
		result[i] = result[i + 1] = result[i + 2] = (unsigned char)(255 * (in[mapIndex] - min) / (max - min));
		result[i + 3] = 255;
	}

	return result;
}
